#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CEIL(a,b) ((a) + (b-1)) / (b)
#define cuda_check(err) _cudaCheck(err, __FILE__, __LINE__)
void _cudaCheck(hipError_t err, const char* file, int line){
    if (err != hipSuccess){
        printf("Cuda ERROR at file %s(line %d) : \n%s\n", file, line, hipGetErrorString(err));
    }
    return;
}

__global__ void sgemv_k64(float* A, float* x, float*y, const int M, const int K){
    int laneID = threadIdx.x % warpSize;    //线程的索引    0 - 31
    int warpID = threadIdx.x / warpSize;    //线程束索引    0 - 1
    int row = blockIdx.x;

    if (row < M){
        float res = 0.0f;
        int iteration = CEIL(K, (2 * warpSize));

        for (int i = 0; i < iteration; i++){
            int col = i * 2 * warpSize + warpID * warpSize + laneID;
            if (col < K) {
                res += A[row * K + col] * x[col];
            }
        }   
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            res += __shfl_down_sync(0xFFFFFFFF, res, offset);
        }
        if (laneID == 0) {
            atomicAdd(&y[row], res);
        }
    }
}

int main(){
    size_t M = 10;
    size_t K = 1280;

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_x = sizeof(float) * K;
    size_t bytes_y = sizeof(float) * M;
    float* h_A  = (float*)malloc(bytes_A);
    float* h_x  = (float*)malloc(bytes_x);
    float* h_y  = (float*)malloc(bytes_y);

    float* d_A;
    float* d_x;
    float* d_y;

    cuda_check(hipMalloc(&d_A, bytes_A));
    cuda_check(hipMalloc(&d_x, bytes_x));
    cuda_check(hipMalloc(&d_y, bytes_y));

    for (int i = 0; i < M * K; i++){
        h_A[i] = float(i / K) + 1;
    }
    for (int i = 0; i < K; i++){
        h_x[i] = 1.0f;
    }
    memset(h_y, 0, bytes_y);

    cuda_check(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    cuda_check(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));

    int block_size = 64;
    int grid_size = M;
    sgemv_k64<<<grid_size, block_size>>>(d_A, d_x, d_y, M, K);

    cuda_check(hipMemcpy( h_y, d_y, bytes_y, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    for (int i = 0;i < M; i ++){
        printf("ans = %f\n", h_y[i]);
    }

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(h_A);
    hipFree(h_x);
    hipFree(h_y);
    return 0;
}